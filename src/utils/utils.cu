#include "utils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstring>

namespace waterSim::utils{
    [[maybe_unused]] bool transferCtxInit(transferCtx& ctx, size_t ptrSize){
        ctx.ptrSize = ptrSize;
        hipError_t err = hipHostMalloc(&ctx.pageLockedHostPtr1, ptrSize);
        if(err != hipSuccess){
            return false;
        }
        err = hipHostMalloc(&ctx.pageLockedHostPtr2, ptrSize);
        if(err != hipSuccess){
            hipFree(ctx.pageLockedHostPtr1);
            return false;
        }
        return true;
    }

    [[maybe_unused]] bool transferCtxDestroy(transferCtx& ctx){
        hipError_t err = hipHostFree(ctx.pageLockedHostPtr1);
        if(err != hipSuccess){
            return false;
        }
        err = hipHostFree(ctx.pageLockedHostPtr2);
        if(err != hipSuccess){
            return false;
        }
        return true;
    }

    [[maybe_unused]] bool fastCudaTransferToDevice(transferCtx& ctx, void* hostPtr, void* devicePtr, size_t size){
        size_t transferred = 0;
        bool isPtr1 = true;
        while (transferred < size){
            size_t toTransfer = ctx.ptrSize;
            if(transferred + toTransfer > size){
                toTransfer = size - transferred;
            }
            if(isPtr1){
                hipDeviceSynchronize();
                std::memcpy(ctx.pageLockedHostPtr1, (std::byte*)hostPtr + transferred, toTransfer);
                hipError_t err = hipMemcpyAsync(devicePtr, ctx.pageLockedHostPtr1, toTransfer, hipMemcpyHostToDevice);
                if(err != hipSuccess){
                    return false;
                }
                transferred += toTransfer;
                isPtr1 = false;
                continue;
            }
            std::memcpy(ctx.pageLockedHostPtr2, (std::byte*)hostPtr + transferred, toTransfer);
            hipError_t err = hipMemcpyAsync(devicePtr, ctx.pageLockedHostPtr2, toTransfer, hipMemcpyHostToDevice);
            if(err != hipSuccess){
                return false;
            }
            transferred += toTransfer;
            isPtr1 = true;
        }
        hipDeviceSynchronize();
        return true;
    }

    [[maybe_unused]] bool fastCudaTransferToHost(transferCtx& ctx, void* devicePtr, void* hostPtr, size_t size){
        size_t transferred = 0;
        bool isPtr1 = true;
        while (transferred < size){
            size_t toTransfer = ctx.ptrSize;
            if(transferred + toTransfer > size){
                toTransfer = size - transferred;
            }
            if(isPtr1){
                hipDeviceSynchronize();
                hipError_t err = hipMemcpyAsync(ctx.pageLockedHostPtr1, devicePtr, toTransfer, hipMemcpyDeviceToHost);
                if(err != hipSuccess){
                    return false;
                }
                std::memcpy((std::byte*)hostPtr + transferred, ctx.pageLockedHostPtr1, toTransfer);
                transferred += toTransfer;
                isPtr1 = false;
                continue;
            }
            hipError_t err = hipMemcpyAsync(ctx.pageLockedHostPtr2, devicePtr, toTransfer, hipMemcpyDeviceToHost);
            if(err != hipSuccess){
                return false;
            }
            std::memcpy((std::byte*)hostPtr + transferred, ctx.pageLockedHostPtr2, toTransfer);
            transferred += toTransfer;
            isPtr1 = true;
        }
        hipDeviceSynchronize();
        return true;
    }
}