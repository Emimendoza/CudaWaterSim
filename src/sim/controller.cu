#include "controller.cuh"
#include "sim.cuh"
#include <algorithm>

waterSim::sim::controller::controller(size_t pointCount, float radius) {
    this -> pointCount = pointCount;
    pointsPosHostActive = new vec3[pointCount];
    hipHostMalloc(&pointsHost, sizeof(point) * pointCount);
    hipMalloc(&pointsDevice, sizeof(point) * pointCount);
    for (size_t i = 0; i < pointCount; i++){
        pointsHost[i] = point(radius);
    }
}

waterSim::sim::controller::~controller() {
    delete[] pointsPosHostActive;
    hipHostFree(pointsHost);
    hipFree(pointsDevice);
    hipFree(modifiersDevice);
}

void waterSim::sim::controller::addModifier(waterSim::sim::modifierI *m) {
    modifiersHost.push_back(m);
    if (modifierArraySize < modifiersHost.size()){
        modifierArraySize = modifiersHost.size()*2;
        hipFree(modifiersDevice);
        hipMalloc(&modifiersDevice, sizeof(modifierI*) * modifierArraySize);
    }
    hipMemcpy(modifiersDevice, modifiersHost.data(), sizeof(modifierI*) * modifiersHost.size(), hipMemcpyHostToDevice);
}

void waterSim::sim::controller::removeModifier(waterSim::sim::modifierI *m) {
    modifiersHost.erase(std::remove(modifiersHost.begin(), modifiersHost.end(), m), modifiersHost.end());
    hipMemcpy(modifiersDevice, modifiersHost.data(), sizeof(modifierI*) * modifiersHost.size(), hipMemcpyHostToDevice);
}

void waterSim::sim::controller::syncDeviceToHost() {
    hipMemcpy(pointsHost, pointsDevice, sizeof(point) * pointCount, hipMemcpyDeviceToHost);
}

void waterSim::sim::controller::syncHostToDevice() {
    hipMemcpy(pointsDevice, pointsHost, sizeof(point) * pointCount, hipMemcpyHostToDevice);
}

void waterSim::sim::controller::step() {
    runModifiers();
    runGravity();
    runCollision();
    updateGraphics();
}

[[noreturn]] void waterSim::sim::controller::mainLoop() {
    while (true){ // TODO: add exit condition
        step();
    }
}

void waterSim::sim::controller::runModifiers() {
    modifyPoints<<<(pointCount + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(pointsDevice, modifiersDevice, modifiersHost.size(), pointCount);
}